#include "hip/hip_runtime.h"
#include <iostream>
#include "math.h"
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <time.h>
using namespace std;
#define ITERATIONS 70000


enum corner_pixel {INSIDE_MASK, BOUNDRY, OUTSIDE};

void convert_layered_to_interleaved(float *aOut, const float *aIn, int w, int h, int nc)
{
	if (nc==1) { memcpy(aOut, aIn, w*h*sizeof(float)); return; }
    size_t nOmega = (size_t)w*h;
    for (int y=0; y<h; y++)
    {
        for (int x=0; x<w; x++)
        {
            for (int c=0; c<nc; c++)
            {
                aOut[(nc-1-c) + nc*(x + (size_t)w*y)] = aIn[x + (size_t)w*y + nOmega*c];
            }
        }
    }
}

void convert_interleaved_to_layered(float *aOut, const float *aIn, int w, int h, int nc)
{
	if (nc==1) { memcpy(aOut, aIn, w*h*sizeof(float)); return; }
    size_t nOmega = (size_t)w*h;
    for (int y=0; y<h; y++)
    {
        for (int x=0; x<w; x++)
        {
            for (int c=0; c<nc; c++)
            {
                aOut[x + (size_t)w*y + nOmega*c] = aIn[(nc-1-c) + nc*(x + (size_t)w*y)];
            }
        }
    }
}


void calculate_boundbox(int target_w, int target_h, int target_nc, int *boundryPixelArray, int *boundBoxMinX, int *boundBoxMinY, int *boundBoxMaxX, int *boundBoxMaxY){
    *boundBoxMaxY = INT32_MIN;
    *boundBoxMaxX = INT32_MIN;
    *boundBoxMinY = INT32_MAX;
    *boundBoxMinX = INT32_MAX;

    for(int channel = 0; channel < target_nc; channel++){
        for(int y = 0; y < target_h; y++){
            for(int x = 0; x < target_w; x++){
                int id = x + y*target_w + channel * target_w * target_h;
                if(boundryPixelArray[id]==BOUNDRY){
                    if(x<*boundBoxMinX){
                        *boundBoxMinX = x;
                    }
                    if(x>*boundBoxMaxX){
                        *boundBoxMaxX = x;
                    }
                    if(y<*boundBoxMinY){
                        *boundBoxMinY = y;
                    }
                    if(y>*boundBoxMaxY){
                        *boundBoxMaxY = y;
                    }
                    
                }
            }
        }
    }

}

void extract_boundary(float *maskIn, int *boundryPixelArray, int source_nchannel, int source_width, int source_height){
    for(int channel = 0; channel < source_nchannel; channel++){
        for(int y = 0; y < source_height; y++){
            for(int x = 0; x < source_width; x++){
                int id = x + y*source_width + channel * source_width * source_height;
                
                // if(x==0 && y==0 && maskIn[id]) boundryPixelArray[id]=CORNER_PIXEL_0_0;
                // else if(x==0 && y==source_height-1 && maskIn[id]) boundryPixelArray[id]=CORNER_PIXEL_0_H;
                // else if(x==source_width-1 && y==0 && maskIn[id]) boundryPixelArray[id]=CORNER_PIXEL_W_0;
                // else if(x==source_width-1 && y==source_height-1 && maskIn[id]) boundryPixelArray[id]=CORNER_PIXEL_W_H;
                // else if(x==0 && y < source_height-1 && maskIn[id]) boundryPixelArray[id]=EDGE_PIXEL_LEFT;
                // else if(x==source_width-1 && y < source_height-1 && maskIn[id]) boundryPixelArray[id]=EDGE_PIXEL_RIGHT;
                // else if(x < source_width-1 && y==0 && maskIn[id]) boundryPixelArray[id]=EDGE_PIXEL_DOWN;
                // else if(x < source_width-1 && y==source_height-1 && maskIn[id]) boundryPixelArray[id]=EDGE_PIXEL_UP;
                if(x==0 && y==0 && maskIn[id]){
                    boundryPixelArray[id]=OUTSIDE;
                }
                else if(x==0 && y==source_height-1 && maskIn[id]){
                    boundryPixelArray[id]=OUTSIDE;
                }
                else if(x==source_width-1 && y==0 && maskIn[id]){
                    boundryPixelArray[id]=OUTSIDE;
                }
                else if(x==source_width-1 && y==source_height-1 && maskIn[id]){
                    boundryPixelArray[id]=OUTSIDE;
                }
                else if(x==0 && y < source_height-1 && maskIn[id]){
                    boundryPixelArray[id]=OUTSIDE;
                }
                else if(x==source_width-1 && y < source_height-1 && maskIn[id]){
                    boundryPixelArray[id]=OUTSIDE;
                }
                else if(x < source_width-1 && y==0 && maskIn[id]){
                    boundryPixelArray[id]=OUTSIDE;
                }
                else if(x < source_width-1 && y==source_height-1 && maskIn[id]){
                    boundryPixelArray[id]=OUTSIDE;
                }
                else{
                    int id_right = x+1 + y*source_width + channel * source_width * source_height;
                    int id_left = x-1 + y*source_width + channel * source_width * source_height;
                    int id_up = x + (y+1)*source_width + channel * source_width * source_height;
                    int id_down = x + (y-1)*source_width + channel * source_width * source_height;

                    if(maskIn[id]>=0.5&&maskIn[id_right]>=0.5&&maskIn[id_left]>=0.5&&maskIn[id_up]>=0.5&&maskIn[id_down]>=0.5){
                        boundryPixelArray[id] = INSIDE_MASK;
                    }
                    else if(maskIn[id]){
                        boundryPixelArray[id] = BOUNDRY;
                    }
                    else{
                        boundryPixelArray[id] = OUTSIDE;
                    }
                }
            }
        }
    }
}

void merge_without_blend(float *srcimg, float *targetimg, float *outimg, int *boundary_array,int source_nchannel, int source_width, int source_height){
    for(int channel = 0; channel < source_nchannel; channel++){
        for(int y = 0; y < source_height; y++){
            for(int x = 0; x < source_width; x++){
                int id = x + y*source_width + channel * source_width * source_height;
                if(boundary_array[id] == INSIDE_MASK){
                    outimg[id] = targetimg[id];
                }
                else{
                    outimg[id] = srcimg[id];
                }
            }
        }
    }
}

void poisson_jacobi(float *targetimg, float *outimg, 
    int *boundary_array,int c, int w, 
    int h, int boundBoxMinX, int boundBoxMaxX, 
    int boundBoxMinY, int boundBoxMaxY){
    for(int i=0; i<ITERATIONS; i++){
        //printf("%d iteration\n", i);
        for(int channel = 0; channel < c; channel++){
            for(int y = boundBoxMinY; y <= boundBoxMaxY; y++){
                for(int x = boundBoxMinX; x <= boundBoxMaxX; x++){
                    int id = x + y*w + channel * w * h;
                    int idx_nextX = x+1 + w*y +w*h*channel;
                    int idx_prevX = x-1 + w*y + w*h*channel;
                    int idx_nextY = x + w*(y+1) +w*h*channel;
                    int idx_prevY = x + w*(y-1) +w*h*channel;
                    //printf("id: %d, idx_nextX: %d, idx_prevX: %d, idx_nextY: %d, idx_prevY: %d\n", id, idx_nextX, idx_prevX, idx_nextY, idx_prevY);
                    if(boundary_array[id] == INSIDE_MASK){
                        double neighbor_target = targetimg[idx_nextY]+targetimg[idx_nextX]+targetimg[idx_prevX]+targetimg[idx_prevY];
                        double neighbor_output = outimg[idx_nextY]+outimg[idx_nextX]+outimg[idx_prevX]+outimg[idx_prevY];
                        outimg[id] = 0.25*(4*targetimg[id]-neighbor_target + neighbor_output);
                    }
                }
            }
        }
    }
}

__global__ void extract_boundary_kernel(float *maskIn, int *boundryPixelArray, int source_nchannel, int source_width, int source_height){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    for(int channel = 0; channel < source_nchannel; channel++){
        if(x<source_width && y<source_height){
            int id = x + source_width * y + source_width * source_height * channel;   
            if(x==0 && y==0 && maskIn[id]){
                boundryPixelArray[id]=OUTSIDE;
            }
            else if(x==0 && y==source_height-1 && maskIn[id]){
                boundryPixelArray[id]=OUTSIDE;
            }
            else if(x==source_width-1 && y==0 && maskIn[id]){
                boundryPixelArray[id]=OUTSIDE;
            }
            else if(x==source_width-1 && y==source_height-1 && maskIn[id]){
                boundryPixelArray[id]=OUTSIDE;
            }
            else if(x==0 && y < source_height-1 && maskIn[id]){
                boundryPixelArray[id]=OUTSIDE;
            }
            else if(x==source_width-1 && y < source_height-1 && maskIn[id]){
                boundryPixelArray[id]=OUTSIDE;
            }
            else if(x < source_width-1 && y==0 && maskIn[id]){
                boundryPixelArray[id]=OUTSIDE;
            }
            else if(x < source_width-1 && y==source_height-1 && maskIn[id]){
                boundryPixelArray[id]=OUTSIDE;
            }
            else{
                int id_right = x+1 + y*source_width + channel * source_width * source_height;
                int id_left = x-1 + y*source_width + channel * source_width * source_height;
                int id_up = x + (y+1)*source_width + channel * source_width * source_height;
                int id_down = x + (y-1)*source_width + channel * source_width * source_height;

                if(maskIn[id]>=0.5&&maskIn[id_right]>=0.5&&maskIn[id_left]>=0.5&&maskIn[id_up]>=0.5&&maskIn[id_down]>=0.5){
                    boundryPixelArray[id] = INSIDE_MASK;
                }
                else if(maskIn[id]){
                    boundryPixelArray[id] = BOUNDRY;
                }
                else{
                    boundryPixelArray[id] = OUTSIDE;
                }
            }
        }
    }
}

__global__ void merge_without_blend_kernel(float *srcimg, float *targetimg, float *outimg, int *boundary_array,int source_nchannel, int source_width, int source_height){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    for(int channel = 0; channel < source_nchannel; channel++){
        int id = x + y*source_width + channel * source_width * source_height;
        if(boundary_array[id] == INSIDE_MASK){
            outimg[id] = targetimg[id];
        }
        else{
            outimg[id] = srcimg[id];
        }
    }
}

__global__ void poisson_jacobi_kernel(float *targetimg, float *outimg, 
    int *boundary_array,int c, int w, 
    int h, int boundBoxMinX, int boundBoxMaxX, 
    int boundBoxMinY, int boundBoxMaxY){

    int x = threadIdx.x + blockIdx.x * blockDim.x + boundBoxMinX;
    int y = threadIdx.y + blockIdx.y * blockDim.y + boundBoxMinY;
    for(int channel = 0; channel < c; channel++){
        int id = x + y*w + channel * w * h;
        int idx_nextX = x+1 + w*y +w*h*channel;
        int idx_prevX = x-1 + w*y + w*h*channel;
        int idx_nextY = x + w*(y+1) +w*h*channel;
        int idx_prevY = x + w*(y-1) +w*h*channel;
        //printf("id: %d, idx_nextX: %d, idx_prevX: %d, idx_nextY: %d, idx_prevY: %d\n", id, idx_nextX, idx_prevX, idx_nextY, idx_prevY);
        if(boundary_array[id] == INSIDE_MASK){
            double neighbor_target = targetimg[idx_nextY]+targetimg[idx_nextX]+targetimg[idx_prevX]+targetimg[idx_prevY];
            double neighbor_output = outimg[idx_nextY]+outimg[idx_nextX]+outimg[idx_prevX]+outimg[idx_prevY];
            outimg[id] = 0.25*(4*targetimg[id]-neighbor_target + neighbor_output);
        }
    }

}

void print_cuda_info(){
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);
}


int main(int argc, char **argv)
{
    int iterations=ITERATIONS;

    string source_image = "";
    string mask = "";
    string target_image = "";

    source_image = argv[1];
    cout<<" source_image   : "<<source_image<<endl;
    
    target_image = argv[2];
    cout<<" target_image   : "<<target_image<<endl;
    
    mask = argv[3];
    cout<<" Mask name   : "<<mask <<endl;

    //load source image
    cv::Mat msourceImage = cv::imread(source_image.c_str(), -1);
    if (msourceImage.data == NULL) { cerr << "ERROR: Could not load source image " << source_image << endl; return 1; }
    cv::Mat mmask = cv::imread(mask.c_str(), -1);
    if (mmask.data == NULL) { cerr << "ERROR: Could not load mask image " << mask << endl; return 1; }
    cv::Mat mtargetImage = cv::imread(target_image.c_str(), -1);
    if (mtargetImage.data == NULL) { cerr << "ERROR: Could not load  image " << mask << endl; return 1; }

    msourceImage.convertTo(mSourceImage,CV_32F);
    mtargetImage.convertTo(mTargetImage,CV_32F);
    mmask.convertTo(mmask,CV_32F);

    msourceImage /= 255.f;
    mtargetImage /= 255.f;
    mmask /= 255.f;

    int source_w = msourceImage.cols;         // width
    int source_h = msourceImage.rows;         // height
    int source_nc = msourceImage.channels();  // number of channels
    cout <<endl<<" Source image   : " << source_w << " x " << source_h << " x " <<source_nc<<endl;

    int target_w = mtargetImage.cols;         // width
    int target_h = mtargetImage.rows;         // height
    int target_nc = mtargetImage.channels();  // number of channels
    cout <<endl<<" target image  : " << target_w << " x " << target_h << " x " <<target_nc<<endl;

    int mask_w = mmask.cols;         // width
    int mask_h = mmask.rows;         // height
    int mask_nc = mmask.channels();  // number of channels
    cout <<endl<<" mask          : " << mask_w << " x " << mask_h << " x " <<mask_nc<<endl;

    cv::Mat mOut_seq(source_h,source_w,source_nc);  

    float *srcimgIn  = new float[(size_t)source_w*source_h*source_nc];
    float *maskIn  = new float[(size_t)mask_w*mask_h*mask_nc];
    float *targetimgIn  = new float[(size_t)target_w*target_h*target_nc];

    convert_interleaved_to_layered (srcimgIn, (float*)msourceImage.data, source_w, source_h, source_nc);
    convert_interleaved_to_layered (maskIn, (float*)mmask.data, mask_w, mask_h, mask_nc);
    convert_interleaved_to_layered(targetimgIn, (float*)mtargetImage.data, target_w, target_h, target_nc);

    int *boundryPixelArray_seq = new int[(size_t)target_w*target_h*mOut_seq.channels()];
    float *imgOut_seq = new float[(size_t)target_w*target_h*mOut_seq.channels()];
    int *boundryPixelArray_cuda;
    float *imgOut_cuda;
    float *srcimgIn_cuda;
    float *maskIn_cuda;
    float *targetimgIn_cuda;

    // begin sequential part clocking
    clock_t t1 = clock();
    //get boundary pixel array to indicate which pixel is corner, edge, inside_mask, boundary or just outside
    extract_boundary(maskIn, boundryPixelArray_seq, source_nc, source_w, source_h);
    int boundBoxMinX, boundBoxMinY, boundBoxMaxX, boundBoxMaxY; 
    // calculate the bounding box for reducing unnecessary calculation
    calculate_boundbox(target_w, target_h, target_nc, boundryPixelArray_seq, &boundBoxMinX, &boundBoxMinY, &boundBoxMaxX, &boundBoxMaxY);
    merge_without_blend(srcimgIn, targetimgIn, imgOut_seq, boundryPixelArray_seq, source_nc, source_w, source_h);
    poisson_jacobi(targetimgIn, imgOut_seq, boundryPixelArray_seq, source_nc, source_w, source_h, boundBoxMinX, boundBoxMaxX, boundBoxMinY, boundBoxMaxY);
    
    clock_t sequential_time = clock()-t1;
    cout << "time cost for CPU: "<<sequential_time * 1.0 / CLOCKS_PER_SEC * 1000 << endl;
    convert_layered_to_interleaved((float*)mOut_seq.data, imgOut_seq, source_w, source_h, source_nc);
    cv::imwrite("FinalImage_sequential.jpg",mOut_seq*255.f);

    /*-------------------cuda------------------*/
    clock_t t2 = clock();
    hipMalloc(&boundryPixelArray_cuda, source_w*source_h*source_nc * sizeof(int));
    hipMalloc(&imgOut_cuda, source_w*source_h*source_nc * sizeof(double));
    hipMalloc(&srcimgIn_cuda, source_w*source_h*source_nc * sizeof(double));
    hipMalloc(&maskIn_cuda, mask_w*mask_h*mask_nc * sizeof(double));
    hipMalloc(&targetimgIn_cuda, target_w*target_h*target_nc * sizeof(double));
    
    hipMemcpy(srcimgIn_cuda, srcimgIn, source_w*source_h*source_nc * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(maskIn_cuda, maskIn, mask_w*mask_h*mask_nc * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(targetimgIn_cuda, targetimgIn, target_w*target_h*target_nc * sizeof(double), hipMemcpyHostToDevice);

    dim3 block_image(30,4,1);
    dim3 grid_image = dim3((source_w+block_image.x-1)/block_image.x, (source_h+block_image.y-1)/block_image.y, 1);
    extract_boundary_kernel<<<grid_image, block_image>>> (maskIn_cuda, boundryPixelArray_cuda, source_nc, source_w, source_h);
    hipMemcpy(boundryPixelArray_seq, boundryPixelArray_cuda, target_w*target_h*target_nc * sizeof(int), hipMemcpyDeviceToHost);
    calculate_boundbox(target_w, target_h, target_nc, boundryPixelArray_seq, &boundBoxMinX, &boundBoxMinY, &boundBoxMaxX, &boundBoxMaxY);
    printf("boundBoxMinX: %d, boundBoxMinY: %d, boundBoxMaxX: %d, boundBoxMaxY: %d\n",boundBoxMinX, boundBoxMinY, boundBoxMaxX, boundBoxMaxY);

    dim3 block_target(30,4,1);
    dim3 grid_target = dim3((boundBoxMaxX-boundBoxMinX+block_target.x)/block_target.x, (boundBoxMaxY-boundBoxMinY+block_target.y)/block_target.y, 1);
    merge_without_blend_kernel<<<grid_image, block_image>>>(srcimgIn_cuda, targetimgIn_cuda, imgOut_cuda, boundryPixelArray_cuda, source_nc, source_w, source_h);

    for(int i=0; i<ITERATIONS; i++){
        poisson_jacobi_kernel<<<grid_target, block_target>>>(targetimgIn_cuda, imgOut_cuda, boundryPixelArray_cuda, source_nc, source_w, source_h, boundBoxMinX, boundBoxMaxX, boundBoxMinY, boundBoxMaxY);
        hipDeviceSynchronize();
    }
    
    hipMemcpy(imgOut_seq, imgOut_cuda, source_w*source_h*source_nc * sizeof(float), hipMemcpyDeviceToHost);
    
    clock_t cuda_time = clock()-t2;
    cout << "time cost for GPU: "<<cuda_time * 1.0 / CLOCKS_PER_SEC * 1000 << endl;
    cout << "speedup for cuda: "<<(sequential_time * 1.0 / CLOCKS_PER_SEC * 1000)/(cuda_time * 1.0 / CLOCKS_PER_SEC * 1000)<<endl;
    print_cuda_info();
    convert_layered_to_interleaved((float*)mOut_seq.data, imgOut_seq, source_w, source_h, source_nc);
    cv::imwrite("FinalImage_cuda.jpg",mOut_seq*255.f);

    hipFree(boundryPixelArray_cuda);
    hipFree(targetimgIn_cuda);
    hipFree(srcimgIn_cuda);
    hipFree(maskIn_cuda);
    hipFree(imgOut_cuda);

    free(srcimgIn);
    free(maskIn);
    free(targetimgIn);
    free(boundryPixelArray_seq);
    free(imgOut_seq);
} 