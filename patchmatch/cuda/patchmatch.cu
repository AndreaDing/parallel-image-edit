#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#include "patchmatch.h"

using namespace std;

#define N_CHANNELS 4

__device__ __inline__ float square(float x) { return x * x; }

__device__ __inline__ int get_max(int x,int y) { return (x > y) ? x : y; }

__device__ __inline__ int get_min(int x,int y) { return (x <= y) ? x : y; }

__device__ __inline__ int get_pidx(int y, int x, int w) { return y * w + x; }

__device__ __inline__ int get_cidx(int y, int x, int w, int c) 
{ 
    return (y * w + x) * N_CHANNELS + c; 
}

__device__ __inline__ 
void init_rand(hiprandState *state) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    hiprand_init(i, j, 0, state);
}

__device__ __inline__ 
float get_rand(hiprandState *state)
{
	return hiprand_uniform(state);
}

__device__ __inline__
float sum_squared_diff(float *fpixel, float *spixel)
{
    float dist = sqrt(
        square(fpixel[0] - spixel[0]) +
        square(fpixel[1] - spixel[1]) +
        square(fpixel[2] - spixel[2])
    );
    return dist;
}

__device__ __inline__
float sum_absolute_diff(float *fpixel, float *spixel)
{
    float dist = sqrt(
        abs(fpixel[0] - spixel[0]) +
        abs(fpixel[1] - spixel[1]) +
        abs(fpixel[2] - spixel[2])
    );
    return dist;
}

__device__ __inline__
float patch_distance(float *first, float *second, 
    int fx, int fy, int sx, int sy, 
    int height, int width, int half_patch)
{
    float dist = 0;
    for (int j = -half_patch; j <= half_patch; j++) {
        for (int i = -half_patch; i <= half_patch; i++) {
            int fx1 = get_min(width - 1, get_max(0, fx + i));
            int fy1 = get_min(height - 1, get_max(0, fy + i));
            float *fpixel = first + get_pidx(fy1, fx1, width) * N_CHANNELS;

            int sx1 = get_min(width - 1, get_max(0, sx + i));
            int sy1 = get_min(height - 1, get_max(0, sy + i));
            float *spixel = second + get_pidx(sy1, sx1, width) * N_CHANNELS;

            dist += sum_squared_diff(fpixel, spixel);
        }
    }
    return dist;
}

__device__
void init_random_map(float *first, float *second, map_t *map, 
    int height, int width, int half_patch)
{
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (y >= height || x >= width) return;

    hiprandState state;
    init_rand(&state);

    int rx = (int)(get_rand(&state) * width) % width;
    int ry = (int)(get_rand(&state) * height) % height;
    int idx = y * width + x;

    map[idx].x = rx;
    map[idx].y = ry;
    map[idx].dist = patch_distance(first, second, x, y, rx, ry, 
        height, width, half_patch);  
}

__device__
void nn_search(float *first, float *second, map_t *curMap, 
    int height, int width, int half_patch)
{
    // int search_radius = get_min(MAX_SEARCH_RADIUS, get_min(width, height));
    int search_radius = get_max(width, height);

    hiprandState state;
    init_rand(&state);

    int fy = threadIdx.y + blockIdx.y * blockDim.y;
    int fx = threadIdx.x + blockIdx.x * blockDim.x;
    if (fy >= height || fx >= width) return;

    int f = (fy * width) + fx;
    int best_x = curMap[f].x; 
    int best_y = curMap[f].y; 
    float best_dist = curMap[f].dist;

    // propagate
    if (fx > 0) {
        // find neighbor's patch
        int pf = f - 1;
        int px = curMap[pf].x + 1;
        int py = curMap[pf].y;
        
        if (px < width) { 
            float dist = patch_distance(first, second, fx, fy, px, py, height, width, half_patch);
            
            if (dist < best_dist) {
                best_x = px; 
                best_y = py;
                best_dist = dist;
            }
        }
    }

    if (fy > 0) {
        // find neighbor's patch
        int pf = f - width;
        int px = curMap[pf].x;
        int py = curMap[pf].y + 1;
        
        if (py < height) { 
            float dist = patch_distance(first, second, fx, fy, px, py, height, width, half_patch);
            
            if (dist < best_dist) {
                best_x = px; 
                best_y = py;
                best_dist = dist;
            }
        }
    }

    // random search
    for (int radius = search_radius; radius >= 1; radius /= 2) {
        int xmin = get_max(best_x - radius, 0);
        int xmax = get_min(best_x + radius, width);
        int ymin = get_max(best_y - radius, 0);
        int ymax = get_min(best_y + radius, height);
        int xlen = (xmax - xmin);
        int ylen = (ymax - ymin);
        int rx = (int)(get_rand(&state) * xlen) % xlen + xmin;
        int ry = (int)(get_rand(&state) * ylen) % ylen + ymin;

        float dist = patch_distance(first, second, fx, fy, rx, ry, height, width, half_patch);

        if (dist < best_dist) {
            best_x = rx;
            best_y = ry;
            best_dist = dist;
        }
    }
    
    curMap[f].x = best_x;
    curMap[f].y = best_y;
    curMap[f].dist = best_dist;
        
}

__device__
void nn_map(float *src, float *dst, map_t *map, 
    int height, int width)
{
    int dy = threadIdx.y + blockIdx.y * blockDim.y;
    int dx = threadIdx.x + blockIdx.x * blockDim.x;
    if (dy >= height || dx >= width) return;
    int idx = dy * width + dx;

    if (map[idx].x < 0 || map[idx].x >= width ||
        map[idx].y < 0 || map[idx].y >= height) {
        return;
    }
    else {
        int midx = get_pidx(map[idx].y, map[idx].x, width);
        dst[idx * N_CHANNELS + 0] = src[midx * N_CHANNELS + 0];
        dst[idx * N_CHANNELS + 1] = src[midx * N_CHANNELS + 1];
        dst[idx * N_CHANNELS + 2] = src[midx * N_CHANNELS + 2];
    } 
}

__device__
void nn_map_average(float *src, float *dst, map_t *map, 
    int height, int width, int half_patch)
{
    int dy = threadIdx.y + blockIdx.y * blockDim.y;
    int dx = threadIdx.x + blockIdx.x * blockDim.x;
    if (dy >= height || dx >= width) return;

    int fy_min = get_max(dy - half_patch, 0);
    int fy_max = get_min(dy + half_patch, height - 1);
    int fy_len = fy_max - fy_min + 1;

    int fx_min = get_max(dx - half_patch, 0);
    int fx_max = get_min(dx + half_patch, width - 1);
    int fx_len = fx_max - fx_min + 1;

    int pixel_sums[3];
    pixel_sums[0] = pixel_sums[1] = pixel_sums[2] = 0;
    
    for (int fy = fy_min; fy <= fy_max; fy++) {
        for (int fx = fx_min; fx <= fx_max; fx++) {
            int f = fy * width + fx;
            int px = map[f].x;
            int py = map[f].y;

            float *spixel = src + get_pidx(py, px, width) * N_CHANNELS;
            pixel_sums[0] += spixel[0];
            pixel_sums[1] += spixel[1];
            pixel_sums[2] += spixel[2];
        }
    }

    int num_pixels = fy_len * fx_len;

    float *dpixel = dst + get_pidx(dy, dx, width) * N_CHANNELS;
    dpixel[0] = pixel_sums[0] / num_pixels;
    dpixel[1] = pixel_sums[1] / num_pixels;
    dpixel[2] = pixel_sums[2] / num_pixels;   
}

__global__ void patchmatch_kernel(float *src, float *dst, map_t *curMap, 
    int height, int width, int half_patch)
{
    init_random_map(dst, src, curMap, height, width, half_patch);
    __syncthreads();

    for (int i = 1; i <= NUM_ITERATIONS; i++) {
        nn_search(dst, src, curMap, height, width, half_patch);
        __syncthreads();
    }
    nn_map_average(src, dst, curMap, height, width, half_patch);
}

__global__ void dummy_kernel(float *src, float *dst, 
    int height, int width, int half_patch)
{
    int dy = threadIdx.y + blockIdx.y * blockDim.y;
    int dx = threadIdx.x + blockIdx.x * blockDim.x;

    int idx = dy * width + dx;
    dst[idx * 4] = 0;
    dst[idx * 4 + 1] = 0;
    dst[idx * 4 + 2] = 0;
}

void patchmatch(float *src, float *dst, int height, int width, int half_patch)
{
    int n_pixels = height * width;

    float *d_src, *d_dst;
    map_t *d_map;
    hipMalloc((void **)&d_src, n_pixels * N_CHANNELS * sizeof(float));
    hipMalloc((void **)&d_dst, n_pixels * N_CHANNELS * sizeof(float));
    hipMalloc((void **)&d_map, n_pixels * sizeof(map_t));

    hipMemcpy(d_src, src, n_pixels * N_CHANNELS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dst, dst, n_pixels * N_CHANNELS * sizeof(float), hipMemcpyHostToDevice); 
    
    int blocksize = 32;
    dim3 blockDim(blocksize, blocksize, 1);
    dim3 gridDim(
        (height + blocksize - 1) / blocksize,
        (width + blocksize - 1) / blocksize, 
        1);

    patchmatch_kernel<<<gridDim, blockDim>>>(d_src, d_dst, d_map, height, width, half_patch);
    // dummy_kernel<<<gridDim, blockDim>>>(d_src, d_dst, width, height, half_patch);
    hipDeviceSynchronize();

    hipMemcpy(dst, d_dst, n_pixels * N_CHANNELS * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_map);
    hipFree(d_src);
    hipFree(d_dst);
}