#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <stdlib.h>

#include <hiprand/hiprand_kernel.h>

#include "patchmatch.h"

using namespace cv;
using namespace std;

__device__ __inline__ float square(float x) { return x * x; }

__device__ __inline__ int get_max(int x,int y)
{
    if(x > y) return x;
    return y;
}

__device__ __inline__ int get_min(int x,int y)
{
    if(x < y) return x;
    return y;
}

__device__ __inline__ void init_rand(hiprandState *state) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    hiprand_init(i, j, 0, state);
}

__device__ __inline__ float get_rand(hiprandState *state)
{
	return hiprand_uniform(state);
}

__device__ __inline__
float sum_squared_diff(uchar3 fpixel, uchar3 spixel)
{
    float dist = sqrt(
        square(fpixel.x - spixel.x) +
        square(fpixel.y - spixel.y) +
        square(fpixel.z - spixel.z)
    );
    return dist;
}

__device__ __inline__
float sum_absolute_diff(uchar3 fpixel, uchar3 spixel)
{
    float dist = sqrt(
        abs(fpixel.x - spixel.x) +
        abs(fpixel.y - spixel.y) +
        abs(fpixel.z - spixel.z)
    );
    return dist;
}

__device__
float patch_distance(uchar3 *first, uchar3 *second, 
    int fx, int fy, int sx, int sy, 
    int width, int height, int half_patch)
{
    float dist = 0;
    for (int j = -half_patch; j <= half_patch; j++) {
        for (int i = -half_patch; i <= half_patch; i++) {
            int fx1 = get_min(width - 1, get_max(0, fx + i));
            int fy1 = get_min(height - 1, get_max(0, fy + i));
            int f = fy1 * width + fx1;

            int sx1 = get_min(width - 1, get_max(0, sx + i));
            int sy1 = get_min(height - 1, get_max(0, sy + i));
            int s = sy1 * width + sx1;

            dist += sum_squared_diff(first[f], second[s]);
        }
    }
    return dist;
}

__device__
void init_random_map(uchar3 *first, uchar3 *second, map_t *map, 
    int width, int height, int half_patch)
{
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (y >= height || x >= width) return;

    hiprandState state;
    init_rand(&state);

    int rx = (int)(get_rand(&state) * width) % width;
    int ry = (int)(get_rand(&state) * height) % height;
    int idx = y * width + x;

    map[idx].x = rx;
    map[idx].y = ry;
    map[idx].dist = patch_distance(first, second, x, y, rx, ry, 
        width, height, half_patch);
}

__device__
void nn_search(uchar3 *first, uchar3 *second, map_t *map, 
    int width, int height, int half_patch)
{
    int fy = threadIdx.y + blockIdx.y * blockDim.y;
    int fx = threadIdx.x + blockIdx.x * blockDim.x;
    if (fy >= height || fx >= width) return;

    int search_radius = get_max(width, height);

    hiprandState state;
    init_rand(&state);

    int f = (fy * width) + fx;
    int best_x = map[f].x; 
    int best_y = map[f].y; 
    float best_dist = map[f].dist;

    // propagate
    if (fx > 0) {
        // find neighbor's patch
        int pf = f - 1;
        int px = map[pf].x + 1;
        int py = map[pf].y;
        
        if (px < width) { 
            float dist = patch_distance(first, second, fx, fy, px, py, 
                width, height, half_patch);
            
            if (dist < best_dist) {
                best_x = px; 
                best_y = py;
                best_dist = dist;
            }
        }
    }

    if (fy > 0) {
        // find neighbor's patch
        int pf = f - width;
        int px = map[pf].x;
        int py = map[pf].y + 1;
        
        if (py < height) { 
            float dist = patch_distance(first, second, fx, fy, px, py, 
                width, height, half_patch);
            
            if (dist < best_dist) {
                best_x = px; 
                best_y = py;
                best_dist = dist;
            }
        }
    }

    // random search
    for (int radius = search_radius; radius >= 1; radius /= 2) {
        // pick a random pixel
        int xmin = get_max(best_x - radius, 0);
        int xmax = get_min(best_x + radius, width);
        int ymin = get_max(best_y - radius, 0);
        int ymax = get_min(best_y + radius, height);
        int xlen = (xmax - xmin);
        int ylen = (ymax - ymin);
        int rx = (int)(get_rand(&state) * xlen) % xlen + xmin;
        int ry = (int)(get_rand(&state) * ylen) % ylen + ymin;

        float dist = patch_distance(first, second, fx, fy, rx, ry, 
            width, height, half_patch);

        if (dist < best_dist) {
            best_x = rx;
            best_y = ry;
            best_dist = dist;
        }
    }
    
    map[f].x = best_x;
    map[f].y = best_y;
    map[f].dist = best_dist;
        
}

__device__
void nn_map(uchar3 *src, uchar3 *dst, map_t *map, 
    int width, int height, int half_patch)
{
    int dy = threadIdx.y + blockIdx.y * blockDim.y;
    int dx = threadIdx.x + blockIdx.x * blockDim.x;
    if (dy >= height || dx >= width) return;
    int idx = dy * width + dx;

    if (map[idx].x < 0 || map[idx].x >= width ||
        map[idx].y < 0 || map[idx].y >= height) {
        return;
    }
    else {
        int midx = map[idx].y * width + map[idx].x;
        dst[idx] = src[midx];
    }  
}

__device__
void nn_map_average(uchar3 *src, uchar3 *dst, map_t *map, 
    int width, int height, int half_patch)
{
    int dy = threadIdx.y + blockIdx.y * blockDim.y;
    int dx = threadIdx.x + blockIdx.x * blockDim.x;
    if (dy >= height || dx >= width) return;
    
    int fy_min = get_max(dy - half_patch, 0);
    int fy_max = get_min(dy + half_patch, height - 1);
    int fy_len = fy_max - fy_min + 1;

    int fx_min = get_max(dx - half_patch, 0);
    int fx_max = get_min(dx + half_patch, width - 1);
    int fx_len = fx_max - fx_min + 1;

    int pixel_sums[3];
    pixel_sums[0] = pixel_sums[1] = pixel_sums[2] = 0;
    
    for (int fy = fy_min; fy <= fy_max; fy++) {
        for (int fx = fx_min; fx <= fx_max; fx++) {
            int f = fy * width + fx;
            int px = map[f].x;
            int py = map[f].y;

            int p = py * width + px;
            uchar3 spixel = src[p];
            pixel_sums[0] += spixel.x;
            pixel_sums[1] += spixel.y;
            pixel_sums[2] += spixel.z;
        }
    }

    int d = dy * width + dx;
    int num_pixels = fy_len * fx_len;
    dst[d].x = pixel_sums[0] / num_pixels;
    dst[d].y = pixel_sums[1] / num_pixels;
    dst[d].z = pixel_sums[2] / num_pixels;
}

__global__ 
void init_kernel(uchar3 *dst, uchar3 *src, map_t *map, 
    int width, int height, int half_patch)
{
    dst[0].x = 30;
    dst[0].y = 45;
    dst[0].z = 100;
    printf("1\n");
}

__global__ 
void patchmatch_kernel(uchar3 *dst, uchar3 *src, map_t *map, 
    int width, int height, int half_patch)
{
    init_random_map(dst, src, map, width, height, half_patch);
    __syncthreads();
    for (int i = 1; i <= NUM_ITERATIONS; i++) {
        nn_search(dst, src, map, width, height, half_patch);
        __syncthreads();
    }
    nn_map_average(src, dst, map, width, height, half_patch);
}

void mat_to_uchar3_array(const cv::Mat &mat, uchar3 **arr_ptr)
{
    int ny = mat.rows;
    int nx = mat.cols;
    uchar3 *arr = (uchar3 *) malloc(ny * nx * sizeof(uchar3));

    int idx = 0;
    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            Vec3b pixel = mat.at<Vec3b>(y, x);
            arr[idx].x = pixel[0];
            arr[idx].y = pixel[1];
            arr[idx].z = pixel[2];
            idx++;
        }
    }

    *arr_ptr = arr;
}

void uchar3_array_to_mat(uchar3 *arr, cv::Mat &mat)
{
    int ny = mat.rows;
    int nx = mat.cols;

    int idx = 0;
    for (int y = 0; y < ny; y++) {
        for (int x = 0; x < nx; x++) {
            uchar3 pixel = arr[idx];
            mat.at<Vec3b>(y, x)[0] = pixel.x;
            mat.at<Vec3b>(y, x)[1] = pixel.y;
            mat.at<Vec3b>(y, x)[2] = pixel.z;
            idx++;
        }
    }
}

void print_uchar3(uchar3 *arr, int len)
{
    for (int i = 0; i < len; i++) {
        cout << (int) arr[i].x << "," << (int) arr[i].y << "," << (int) arr[i].z << endl;
    }
}

void print_map(map_t *map, int len)
{
    for (int i = 0; i < len; i++) {
        cout << map[i].x << "," << map[i].y << "," << map[i].dist << endl;
    }
}

void patchmatch(const cv::Mat &srcMat, cv::Mat &dstMat, int half_patch)
{
    if (srcMat.rows != dstMat.rows || srcMat.cols != dstMat.cols) {
        cout << "Error: size not match." << endl;
        return;
    }

    int height = dstMat.rows;
    int width = dstMat.cols;
    int len = height * width;

    uchar3 *src, *dst;
    mat_to_uchar3_array(srcMat, &src);
    mat_to_uchar3_array(dstMat, &dst);
    map_t *map = (map_t *) malloc(height * width * sizeof(map_t));

    cout << "src" << endl;
    print_uchar3(src, 10);
    cout << "dst" << endl;
    print_uchar3(dst, 10);

    uchar3 *d_src, *d_dst;
    map_t *d_map;
    hipMalloc((void **)&d_src, len * sizeof(uchar3));
    hipMalloc((void **)&d_dst, len * sizeof(uchar3));
    hipMalloc((void **)&d_map, len * sizeof(map_t));

    hipMemcpy(d_src, src, len * sizeof(uchar3), hipMemcpyHostToDevice);
    hipMemcpy(d_dst, dst, len * sizeof(uchar3), hipMemcpyHostToDevice); 
    
    int blocksize = 32;
    dim3 blockDim(blocksize, blocksize, 1);
    dim3 gridDim(
        (height + blocksize - 1) / blocksize,
        (width + blocksize - 1) / blocksize, 
        1);

    init_kernel<<<blockDim, gridDim>>>(d_dst, d_src, d_map, width, height, half_patch);
    hipDeviceSynchronize();
    
    hipMemcpy(dst, d_dst, len * sizeof(uchar3), hipMemcpyDeviceToHost);
    hipMemcpy(map, d_map, len * sizeof(map_t), hipMemcpyDeviceToHost);

    cout << "dst" << endl;
    print_uchar3(dst, 10);
    cout << "map" << endl;
    print_map(map, 10);

    patchmatch_kernel<<<blockDim, gridDim>>>(d_dst, d_src, d_map, width, height, half_patch);
    hipDeviceSynchronize();

    hipMemcpy(dst, d_dst, len * sizeof(uchar3), hipMemcpyDeviceToHost);
    hipMemcpy(map, d_map, len * sizeof(map_t), hipMemcpyDeviceToHost);

    cout << "dst" << endl;
    print_uchar3(dst, 10);
    cout << "map" << endl;
    print_map(map, 10);

    uchar3_array_to_mat(dst, dstMat);

    hipFree(d_map);
    hipFree(d_src);
    hipFree(d_dst);

    free(src);
    free(dst);
}